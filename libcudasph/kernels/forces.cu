#include "hip/hip_runtime.h"


__device__ float compute_density_with_grid(
    size_t current_particle_index, const particle* others,
    const simulation_parameters params,
    const precomputed_kernel_values smoothing_terms,
    const unsigned int* grid_cell_particle_list) {
  float density = 0.f;

  uint3 cell_coords =
      get_cell_coords_z_curve(others[current_particle_index].grid_index);

  for (uint z = cell_coords.z - 1; z <= cell_coords.z + 1; ++z) {
    for (uint y = cell_coords.y - 1; y <= cell_coords.y + 1; ++y) {
      for (uint x = cell_coords.x - 1; x <= cell_coords.x + 1; ++x) {
        uint grid_index = get_grid_index_z_curve(x, y, z);
        uint2 indices = get_start_end_indices_for_cell(
            grid_index, grid_cell_particle_list, params);

        for (size_t i = indices.x; i < indices.y; ++i) {
          density += params.particle_mass *
                     poly_6(distance(others[current_particle_index].position,
                                     others[i].position),
                            params.h, smoothing_terms);
        }
      }
    }
  }

  return density;
}

/*float3 compute_internal_forces_with_grid(
    size_t current_particle_index,  const particle* others,
    const simulation_parameters params,
    const precomputed_kernel_values smoothing_terms,
    const unsigned int* grid_cell_particle_list) {
  float3 pressure_term = {0.f, 0.f, 0.f};
  float3 viscosity_term = {0.f, 0.f, 0.f};
  // compute the inward surface normal, it's the gradient of the color field
  float3 normal = {0.f, 0.f, 0.f};
  // also need the color field laplacian
  float color_field_laplacian = 0.f;

  uint3 cell_coords =
      get_cell_coords_z_curve(others[current_particle_index].grid_index);

  for (uint z = cell_coords.z - 1; z <= cell_coords.z + 1; ++z) {
    for (uint y = cell_coords.y - 1; y <= cell_coords.y + 1; ++y) {
      for (uint x = cell_coords.x - 1; x <= cell_coords.x + 1; ++x) {
        uint grid_index = get_grid_index_z_curve(x, y, z);
        uint2 indices = get_start_end_indices_for_cell(
            grid_index, grid_cell_particle_list, params);

        for (size_t i = indices.x; i < indices.y; ++i) {
          if (i != current_particle_index) {
            //[kelager] (4.11)
            pressure_term +=
                (others[i].pressure / powf(others[i].density, 2) +
                 others[current_particle_index].pressure /
                     powf(others[current_particle_index].density, 2)) *
                params.particle_mass *
                spiky_gradient(others[current_particle_index].position -
                                   others[i].position,
                               params.h, smoothing_terms);

            viscosity_term +=
                (others[i].velocity - others[current_particle_index].velocity) *
                (params.particle_mass / others[i].density) *
                viscosity_laplacian(
                    length(others[current_particle_index].position -
                           others[i].position),
                    params.h, smoothing_terms);
          }

          normal += params.particle_mass / others[i].density *
                    poly_6_gradient(others[current_particle_index].position -
                                        others[i].position,
                                    params.h, smoothing_terms);

          color_field_laplacian +=
              params.particle_mass / others[i].density *
              poly_6_laplacian(length(others[current_particle_index].position -
                                      others[i].position),
                               params.h, smoothing_terms);
        }
      }
    }
  }

  float3 sum = (-others[current_particle_index].density * pressure_term) +
               (viscosity_term * params.dynamic_viscosity);

  if (length(normal) > params.surface_tension_threshold) {
    sum += -params.surface_tension * color_field_laplacian * normal /
           length(normal);
  }

  return sum;
}*/
