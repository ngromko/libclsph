#include "hip/hip_runtime.h"

/**
 * @brief      Locates the particle data for a certain grid cell in the cell
 *table.
 *
 * @param[in]  cell_index       The index of the grid cell we are examining.
 * @param[in]  cell_table       A flattened representation of the grid contents
 * @param[in]  params           Contains the simulation parameters
 *
 * @return     The start and finish indexes of the subarray that contains the
 *particles that can be found at cell_index.
 *
 */
__device__ uint2 get_start_end_indices_for_cell(uint cell_index,
                                                const unsigned int* cell_table,
                                                simulation_parameters params) {
  uint2 indices = {
      cell_table[cell_index], (params.grid_cell_count > cell_index + 1)
                                  ? cell_table[cell_index + 1]
                                  : params.particles_count,
  };

  return indices;
}

/**
 * @brief Updates each particle with its position in the grid and fills an array
 *with the number of particles contained in each grid cell
 *
 * @param[in]  particles        Contains all the particle data
 * @param[out] out_particles    Will contain the particle data with the added
 *information
 * @param[in]  params           Contains the simulation parameters
 */
__global__ void locate_in_grid(const particle* particles,
                               particle* out_particles,
                               simulation_parameters params) {
  const size_t current_particle_index = blockIdx.x * blockDim.x + threadIdx.x;
  out_particles[current_particle_index] = particles[current_particle_index];

  uint3 position_in_grid = {0, 0, 0};

  float x_min = params.min_point.x;
  float y_min = params.min_point.y;
  float z_min = params.min_point.z;

  // Grid cells will always have a radius length h
  position_in_grid.x = (uint)(
      (particles[current_particle_index].position.x - x_min) / (params.h * 2));
  position_in_grid.y = (uint)(
      (particles[current_particle_index].position.y - y_min) / (params.h * 2));
  position_in_grid.z = (uint)(
      (particles[current_particle_index].position.z - z_min) / (params.h * 2));

  uint grid_index = get_grid_index_z_curve(
      position_in_grid.x, position_in_grid.y, position_in_grid.z);

  out_particles[current_particle_index].grid_index = grid_index;
}
